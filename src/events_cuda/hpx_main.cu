#include "hip/hip_runtime.h"
#include <chrono>
#include <deque>
#include <fstream>
#include <iostream>
#include <thread>

#include "HPXSched.h"
#include <fmt/chrono.h>
#include <fmt/format.h>
#include <hpx/wrap_main.hpp>

// using Mtrx = Eigen::Matrix<double, 10, 10>;

#include "CUDAMtrx.h"
using Mtrx = CUDAMtrx<100>;
constexpr int n_evts_per_block = 30;
using namespace std::chrono_literals;

// setup later
hipblasHandle_t cublas_hndl;
hiprandGenerator_t curand_gen;

template <class R, class P> void busy_wait(std::chrono::duration<R, P> time) {
    // Busy waits for a given length of time.
    auto start = std::chrono::steady_clock::now();
    while (std::chrono::steady_clock::now() - start < time) {
    }
}

Mtrx* make_mtrx(long long x) {
    Mtrx* mtrx = new Mtrx(x);
    return mtrx;
}

long long plus(Mtrx* x, Mtrx* y) {
    float ans = (*x + *y).norm();
    return ans;
}

long long scal_plus(long long x, long long y) {
    return x + y;
}

long long times(Mtrx* x, Mtrx* y) {
    float ans = (*x * *y).norm();
    return ans;
}

long long square(long long x) {
    return x * x;
}

long long cube(long long x) {
    return x * x * x;
}

sch::Sched scheduler{
      sch::Define("Matrix X"_s, hana::make_tuple("X"_in), make_mtrx),
      sch::Define("Matrix Y"_s, hana::make_tuple("Y"_in), make_mtrx),
      sch::Define("Cube Plus"_s, hana::make_tuple("Y plus X"_s), cube),
      sch::Define("Cube Times"_s, hana::make_tuple("Y times X"_s), cube),
      sch::Define("Y plus X"_s, hana::make_tuple("Matrix X"_s, "Matrix Y"_s), plus),
      sch::Define("Y times X"_s, hana::make_tuple("Matrix X"_s, "Matrix Y"_s), times),
      sch::Define("Square Plus"_s, hana::make_tuple("Y plus X"_s), square),
      sch::Define("Square Times"_s, hana::make_tuple("Y times X"_s), square),
      sch::Define("Add Squares"_s, hana::make_tuple("Square Plus"_s, "Square Times"_s), scal_plus)};

struct EvtCtx : public decltype(scheduler)::ECBase {
    long long X = 5;
    long long Y = 10;
};
BOOST_HANA_ADAPT_STRUCT(EvtCtx, X, Y);

int main(int argc, char* argv[]) {
    if (argc != 2) {
        fmt::print("Usage: {} input_file\n", argv[0]);
    }

    // setup CUDA
    setup();
    std::ifstream in{argv[1]};
    std::deque<EvtCtx> evts{};
    std::deque<hpx::shared_future<long long>> outputs{};
    std::deque<hpx::future<void>> cleanups{};

    long long n_evts = 0;
    std::chrono::duration<double, std::milli> total_time = 0ms;
    while (in.good()) {
        EvtCtx ec_template{};
        in >> ec_template.X >> ec_template.Y;
        if (!in.good()) {
            break;
        }
        auto start_tm = std::chrono::steady_clock::now();
        for (int i = 0; i < n_evts_per_block; ++i) {
            EvtCtx& ec = evts.emplace_back();
            ec = ec_template;
            auto& final_ans = scheduler.retrieve(ec, "Add Squares"_s);
            bool success = scheduler.schedule(ec);
            outputs.push_back(*final_ans);
            cleanups.emplace_back(final_ans->then(scheduler.cleanup(ec)));
            n_evts++;
        }
        auto this_time = std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(
              std::chrono::steady_clock::now() - start_tm);
        total_time += this_time;
        fmt::print("Took {} to schedule {} events\n", this_time, n_evts_per_block);
    }
    fmt::print("Waiting for all events\n");
    auto start_tm = std::chrono::steady_clock::now();
    hpx::wait_all(outputs.begin(), outputs.end());
    auto extra_tm = std::chrono::duration_cast<std::chrono::duration<float, std::milli>>(
          std::chrono::steady_clock::now() - start_tm);
    fmt::print("Took {} ({} average) extra waiting for all events\n", extra_tm, extra_tm / n_evts);
    fmt::print("Took {} total ({} average) scheduling events\n", total_time, total_time / n_evts);
    start_tm = std::chrono::steady_clock::now();
    volatile long long o = 0;
    for (auto&& out : outputs) {
        o = out.get();
    }
    fmt::print("Took {} reading out futures\n",
               std::chrono::duration_cast<std::chrono::duration<float, std::milli>>(
                     std::chrono::steady_clock::now() - start_tm));
    // wait for all cleanup to be done
    hpx::wait_all(cleanups.begin(), cleanups.end());

    // cleanup CUDA
    // teardown();
    return 0;
}
